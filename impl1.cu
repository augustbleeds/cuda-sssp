#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

using namespace std

__global__ void pulling_kernel(vector<initial_vertex> * peeps, int offset, int * anyChange){

	//update me based on my neighbors. Toggle anyChange as needed.
	//offset will tell you who I am.
}

void swap(vector<int>* distance_cur, vector<int>* distance_prev){
	
}

void puller(vector<initial_vertex> * peeps, int blockSize, int blockNum){
	setTime();

	//housekeeping goes here
	//populate distance_prev & distance_cur
	
	for(int i=0; i < peeps->size()-1; i++){
		pulling_kernel<<<blockNum,blockSize>>>(peeps,/*distance_prev*/,/*distance_cur*/);
		if(/*no node is changed*/){
			break;
		} else {
			swap(/*distance_cur*/,/*distance_prev*/)
		}
	}

	//more housekeeping needed

	cout << "Took " << getTime() << "ms.\n";
}
